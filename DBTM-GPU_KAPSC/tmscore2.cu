#include "hip/hip_runtime.h"
#include"head.h"
__device__ double rms[350];
__device__  double rr[350][3][3]= {0};
__device__  double hm[350][3] = {0};
//temporarily
__device__  double tmscore[350]= {0};
__device__  double tmscore2[350]= {0};

__device__  double rp1[350][200][3]= {0};
__device__  double rp2[350][200][3]= {0};

__device__  double dist[65000]= {0};
__device__  double u[350][200][3]= {0};

//__device__  double tdp[6000]= {0};
__device__  double ialignt[200*350]= {0};

__device__  int nalign[350]= {0};

__device__ int iter[350]= {0};

__device__ int nchange[350]= {0};

__device__ int tn=301;

__device__
//__global__


__device__
//__global__
void
calculate_tm(double d0,double d,int pl,double *p0,double *p1)
{

   
    int tx=threadIdx.x+blockIdx.x*blockDim.x;
    int ty=threadIdx.y;

    if(tx<tn)
    {
        double d2=0,dis=0;
        //int nchange=0;
        int k,k2;
        tmscore2[tx]=0.0;

        for(k=0; k<pl; k++)
        {
			for(k2=0; k2<3; k2++)
				u[tx][k][k2]=hm[tx][k2] +(rr[tx][k2][0]*p0[k]+rr[tx][k2][1]*p0[pl+k]+rr[tx][k2][2]*p0[2*pl+k])-\
							   p1[tx*3*pl+k2*pl+k];
			
            dis=u[tx][k][0]*u[tx][k][0]+u[tx][k][1]*u[tx][k][1]+u[tx][k][2]*u[tx][k][2];
            tmscore2[tx]+=1.0/(1.0+dis/d0/d0);
            dist[tx*pl+k]=dis;

        }


        int ncut=0;
        while(ncut<3)
        {
            d2=d*d;
            ncut=0;
            for(k=0; k<pl; k++)
            {
                if(dist[tx*pl+k]<d2) ncut++;
            }
            d+=0.5;
        }

        nchange[tx]=0;

        ncut=0;
        for(k=0; k<pl; k++)
        {
            //	if(ty==0 && tx%3==0)
            //		cuPrintf(" =======%lf   %lf \n",dist[tx*pl+k],d2);
            if(dist[tx*pl+k]<d2)
            {
                if(ncut<nalign[tx] && ialignt[tx*pl+ncut]==k) ncut++;
                else
                {
                    nchange[tx]=1;
                    ialignt[tx*pl+ncut]=k;
                    ncut++;
                }

            }

            __syncthreads();
        }

        __syncthreads();
        //if(ty==0 && tx%3==0)
        //cuPrintf("%lf \n",tmscore[tx]);

        if(tmscore2[tx]/(double)pl>tmscore[tx])
            tmscore[tx]=tmscore2[tx]/(double)pl;


        int n=0;
        for(k=0; k<ncut; k++)
        {
            int m=ialignt[tx*pl+k];
            
		    rp1[tx][n][0]=p0[0*pl+m];
            rp1[tx][n][1]=p0[1*pl+m];
            rp1[tx][n][2]=p0[2*pl+m];
						 
			rp2[tx][n][0]=p1[tx*pl*3+0*pl+m];
            rp2[tx][n][1]=p1[tx*pl*3+1*pl+m];
            rp2[tx][n][2]=p1[tx*pl*3+2*pl+m];
						
			
            n++;

        }

        __syncthreads();
        //if(ty==0&&tx%3==0)
        //	cuPrintf("dddddddd  %d \n",ncut);
        nalign[tx]=ncut;
    }
}


__device__ double d0[350]= {0};
__device__ double d[350]= {0};
__device__ int seed[350]={0};
__device__ int i[350]={0};
__device__  int istart[350]={0};

__global__
void
tmscore_gpu(int pl,double *p0,double *p1,double *tm2)
{

    

    int tx=threadIdx.x+blockIdx.x*blockDim.x;
    int ty=threadIdx.y;


    if(tx<tn)
    {
            d0[tx]=1.24*powf((pl-15),(1.0/3.0))-1.8;
            if(d0[tx]<0.5) d0[tx]=0.5;
            double d0_search=d0[tx];
            if(d0_search>8)d0_search=8;
            if(d0_search<4.5) d0_search=4.5;

            int n_it=20;
            //int n_init_max=6;
            int n_init=0;
            int L_init_min=4;
            int L_ini[6];

            if(pl<4) L_init_min=pl;
            int len=pl;
            int divisor=1;

            while(len>L_init_min && n_init<5)
            {
                L_ini[n_init++]=len;
                divisor*=2;
                len=pl/divisor;
            }
            L_ini[n_init++]=4;
            if(L_ini[n_init-1]>L_init_min) L_ini[n_init++]=L_init_min;

		threadfence();
        for(seed[tx]=0; seed[tx]<n_init; seed[tx]++)
        {
            int L_init=L_ini[seed[tx]];
            //if(ty==0&&tx%3==0)
            //	cuPrintf("L_init ====================== %d \n",L_init);
            for(istart[tx]=0; istart[tx]<=pl-L_init; istart[tx]++)
            {
                int n;
                nalign[tx]=L_init;
                {
                    n=0;
                    for(i[tx]=0; i[tx]<pl; i[tx]++)
                    {
                        if(i[tx]>=istart[tx] && i[tx]<istart[tx]+L_init)
                        {

                            rp1[tx][n][0]=p0[0*pl+i[tx]];
                            rp1[tx][n][1]=p0[1*pl+i[tx]];
                            rp1[tx][n][2]=p0[2*pl+i[tx]];
						 
							rp2[tx][n][0]=p1[tx*pl*3+0*pl+i[tx]];
                            rp2[tx][n][1]=p1[tx*pl*3+1*pl+i[tx]];
                            rp2[tx][n][2]=p1[tx*pl*3+2*pl+i[tx]];
						
                            n++;
                        }
                    }
					__syncthreads();
                }


				Kabsch(*(rp1+tx),*(rp2+tx),nalign[tx],1,&rms[tx],*(hm+tx),*(rr+tx));


				calculate_tm(d0[tx],d0_search-1,pl,p0,p1);
                
				d[tx]=d0_search+1;


                //if(tx%3>1000)
                {
                    for(iter[tx]=0; iter[tx]<n_it&&nchange[tx]; iter[tx]++)
                    {
                
						Kabsch(*(rp1+tx),*(rp2+tx),nalign[tx],1,&rms[tx],*(hm+tx),*(rr+tx));
						calculate_tm(d0[tx],d[tx],pl,p0,p1);
					 }
                }
            }
        }
        tm2[tx]=tmscore[tx];
        __syncthreads();

    }
}

int main(void)
{
    int length;
	int i;
    int size=350*200*3;
    double  *p=(double  *)malloc(size*sizeof(double ));

    double *tm=(double  *)malloc(350*sizeof(double ));

    getnpdb("l1",p,&length);

    double  *dp1; //pdb info
    hipMalloc((void**)&dp1,size*sizeof(double));
    hipMemcpy(dp1,p,size*sizeof(double),hipMemcpyHostToDevice);

    double  *dp0; //pdb info
    hipMalloc((void**)&dp0,3*200*sizeof(double));
    hipMemcpy(dp0,p,3*200*sizeof(double),hipMemcpyHostToDevice);


    double  *dtm;
    hipMalloc((void**)&dtm,350*sizeof(double));
    hipMemset(dtm,0,350*sizeof(double));


    dim3 grid(100,1);
    dim3 block(3,1);

    cudaPrintfInit();
    tmscore_gpu<<<grid,block>>>(140,dp0,dp1,dtm);
    cudaPrintfDisplay(stdout, true);
    cudaPrintfEnd();

    hipMemcpy(tm,dtm,350*sizeof(double),hipMemcpyDeviceToHost);
    
	for(i=0;i<300;i++)
	{
		printf("%lf \n",tm[i]);
	}

    hipFree(dp0);
    hipFree(dp1);
    hipFree(dtm);
    free(tm);
    free(p);

    return 0;
}
