#include "hip/hip_runtime.h"
#include"head.h"
__device__ double rms[350];
__device__  double rr[350][3][3]= {0};
__device__  double hm[350][3] = {0};
//temporarily
__device__  double tmscore[350]= {0};
__device__  double tmscore2[350]= {0};

__device__  double rp1[350][200][3]= {0};
__device__  double rp2[350][200][3]= {0};

__device__  double dist[65000]= {0};
__device__  double u[350][200][3]= {0};

//__device__  double tdp[6000]= {0};
__device__  double ialignt[200*350]= {0};

__device__  int nalign[350]= {0};

__device__ int iter[350]= {0};

__device__ int nchange[350]= {0};

__device__ int tn=301;

__device__
//__global__


__device__
//__global__
void
calculate_tm(double d0,double d,int pl,double *p0,double *p1)
{

    int bidx=blockIdx.x;
    //int tx=threadIdx.x+blockIdx.x*blockDim.x;
    int ty=threadIdx.y;

    if(bidx<tn)
    {
        double d2=0,dis=0;
        //int nchange=0;
        int k,k2;
        tmscore2[bidx]=0.0;

        for(k=0; k<pl; k++)
        {
			for(k2=0; k2<3; k2++)
				u[bidx][k][k2]=hm[bidx][k2] +(rr[bidx][k2][0]*p0[k]+rr[bidx][k2][1]*p0[pl+k]+rr[bidx][k2][2]*p0[2*pl+k])-\
							   p1[bidx*3*pl+k2*pl+k];
			
            dis=u[bidx][k][0]*u[bidx][k][0]+u[bidx][k][1]*u[bidx][k][1]+u[bidx][k][2]*u[bidx][k][2];
            tmscore2[bidx]+=1.0/(1.0+dis/d0/d0);
            dist[bidx*pl+k]=dis;

        }


        int ncut=0;
        while(ncut<3)
        {
            d2=d*d;
            ncut=0;
            for(k=0; k<pl; k++)
            {
                if(dist[bidx*pl+k]<d2) ncut++;
            }
            d+=0.5;
        }

        nchange[bidx]=0;

        ncut=0;
        for(k=0; k<pl; k++)
        {
            //	if(ty==0 && tx%3==0)
            //		cuPrintf(" =======%lf   %lf \n",dist[bidx*pl+k],d2);
            if(dist[bidx*pl+k]<d2)
            {
                if(ncut<nalign[bidx] && ialignt[bidx*pl+ncut]==k) ncut++;
                else
                {
                    nchange[bidx]=1;
                    ialignt[bidx*pl+ncut]=k;
                    ncut++;
                }

            }

            __syncthreads();
        }

        __syncthreads();
        //if(ty==0 && tx%3==0)
        //cuPrintf("%lf \n",tmscore[bidx]);

        if(tmscore2[bidx]/(double)pl>tmscore[bidx])
            tmscore[bidx]=tmscore2[bidx]/(double)pl;


        int n=0;
        for(k=0; k<ncut; k++)
        {
            int m=ialignt[bidx*pl+k];
            
		    rp1[bidx][n][0]=p0[0*pl+m];
            rp1[bidx][n][1]=p0[1*pl+m];
            rp1[bidx][n][2]=p0[2*pl+m];
						 
			rp2[bidx][n][0]=p1[bidx*pl*3+0*pl+m];
            rp2[bidx][n][1]=p1[bidx*pl*3+1*pl+m];
            rp2[bidx][n][2]=p1[bidx*pl*3+2*pl+m];
						
			
            n++;

        }

        __syncthreads();
        //if(ty==0&&tx%3==0)
        //	cuPrintf("dddddddd  %d \n",ncut);
        nalign[bidx]=ncut;
    }
}


__device__ double d0[35]= {0};
__device__ double d[35]= {0};
__device__ int seed[35]={0};
__device__ int i[35]={0};
__device__  int istart[35]={0};

__global__
void
tmscore_gpu(int pl,double *p0,double *p1,double *tm2)
{

    int bidx=blockIdx.x;

    int tx=threadIdx.x+blockIdx.x*blockDim.x;
    int ty=threadIdx.y;


    if(bidx<tn)
    {
            d0[bidx]=1.24*powf((pl-15),(1.0/3.0))-1.8;
            if(d0[bidx]<0.5) d0[bidx]=0.5;
            double d0_search=d0[bidx];
            if(d0_search>8)d0_search=8;
            if(d0_search<4.5) d0_search=4.5;

            int n_it=20;
            //int n_init_max=6;
            int n_init=0;
            int L_init_min=4;
            int L_ini[6];

            if(pl<4) L_init_min=pl;
            int len=pl;
            int divisor=1;

            while(len>L_init_min && n_init<5)
            {
                L_ini[n_init++]=len;
                divisor*=2;
                len=pl/divisor;
            }
            L_ini[n_init++]=4;
            if(L_ini[n_init-1]>L_init_min) L_ini[n_init++]=L_init_min;

		threadfence();
        for(seed[bidx]=0; seed[bidx]<n_init; seed[bidx]++)
        {
            int L_init=L_ini[seed[bidx]];
            //if(ty==0&&tx%3==0)
            //	cuPrintf("L_init ====================== %d \n",L_init);
            for(istart[bidx]=0; istart[bidx]<=pl-L_init; istart[bidx]++)
            {
                int n;
                nalign[bidx]=L_init;
                {
                    n=0;
                    for(i[bidx]=0; i[bidx]<pl; i[bidx]++)
                    {
                        if(i[bidx]>=istart[bidx] && i[bidx]<istart[bidx]+L_init)
                        {

                            rp1[bidx][n][0]=p0[0*pl+i[bidx]];
                            rp1[bidx][n][1]=p0[1*pl+i[bidx]];
                            rp1[bidx][n][2]=p0[2*pl+i[bidx]];
						 
							rp2[bidx][n][0]=p1[bidx*pl*3+0*pl+i[bidx]];
                            rp2[bidx][n][1]=p1[bidx*pl*3+1*pl+i[bidx]];
                            rp2[bidx][n][2]=p1[bidx*pl*3+2*pl+i[bidx]];
						
                            n++;
                        }
                    }
                }


				Kabsch(*(rp1+bidx),*(rp2+bidx),nalign[bidx],1,&rms[bidx],*(hm+bidx),*(rr+bidx));

/*
				int i,j;
				for(i=0;i<3;i++)
				{
					cuPrintf("%lf ",hm[bidx][i]);
					for(j=0;j<3;j++)
					{
						
						cuPrintf("%lf  \n",rr[bidx][i][j]);
					}
				}
*/
				calculate_tm(d0[bidx],d0_search-1,pl,p0,p1);
                
				d[bidx]=d0_search+1;


                //if(tx%3>1000)
                {
                    for(iter[bidx]=0; iter[bidx]<n_it&&nchange[bidx]; iter[bidx]++)
                    {
                
						Kabsch(*(rp1+bidx),*(rp2+bidx),nalign[bidx],1,&rms[bidx],*(hm+bidx),*(rr+bidx));
						calculate_tm(d0[bidx],d[bidx],pl,p0,p1);
					 }
                }
            }
        }
        tm2[bidx]=tmscore[bidx];
        __syncthreads();

    }
}

int main(void)
{
    int length;
	int i;
    int size=350*200*3;
    double  *p=(double  *)malloc(size*sizeof(double ));

    double *tm=(double  *)malloc(350*sizeof(double ));

    getnpdb("l1",p,&length);

    double  *dp1; //pdb info
    hipMalloc((void**)&dp1,size*sizeof(double));
    hipMemcpy(dp1,p,size*sizeof(double),hipMemcpyHostToDevice);

    double  *dp0; //pdb info
    hipMalloc((void**)&dp0,3*200*sizeof(double));
    hipMemcpy(dp0,p,3*200*sizeof(double),hipMemcpyHostToDevice);


    double  *dtm;
    hipMalloc((void**)&dtm,350*sizeof(double));
    hipMemset(dtm,0,350*sizeof(double));


    dim3 grid(100,1);
    dim3 block(1,1);

    cudaPrintfInit();
    tmscore_gpu<<<grid,block>>>(140,dp0,dp1,dtm);
    cudaPrintfDisplay(stdout, true);
    cudaPrintfEnd();

    hipMemcpy(tm,dtm,350*sizeof(double),hipMemcpyDeviceToHost);
    
	for(i=0;i<100;i++)
	{
		printf("%lf \n",tm[i]);
	}

    hipFree(dp0);
    hipFree(dp1);
    hipFree(dtm);
    free(tm);
    free(p);

    return 0;
}
